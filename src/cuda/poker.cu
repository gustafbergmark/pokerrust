#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void p_sum(float *input, int i) {
    int offset = 1;
    for (int d = 64; d > 0; d >>= 1) {
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            input[bi] += input[ai];
        }
        offset *= 2;
    }
    if (i == 0) {
        input[127] = 0;
    }
    for (int d = 1; d < 128; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            float t = input[ai];
            input[ai] = input[bi];
            input[bi] += t;
        }
    }
    __syncthreads();
}

__global__ void gpu_prefix_sum(float *input) {
    __shared__ float temp[128];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        if (i * 11 + b < 1326 && i < 127) {
            temp[i] += input[i * 11 + b];
        }
    }

    __syncthreads();
    p_sum(temp, i);

    float prefix = temp[i];
    for (int b = 0; b < 11; b++) {
        if (i * 11 + b < 1326) {
            float temp = input[i * 11 + b];
            input[i * 11 + b] = prefix;
            prefix += temp;
        }
    }
}

__device__ void cuda_prefix_sum(float *input) {
    __shared__ float temp[128];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp[i] = 0;
    for (int b = 0; b < 11; b++) {
        if (i * 11 + b < 1326 && i < 127) {
            temp[i] += input[i * 11 + b];
        }
    }
    __syncthreads();
    p_sum(temp, i);

    float prefix = temp[i];
    for (int b = 0; b < 11; b++) {
        if (i * 11 + b < 1326) {
            float t = input[i * 11 + b];
            input[i * 11 + b] = prefix;
            prefix += t;
        }
    }
}

// returns the largest value smaller than or equal to val in list
__device__ short lower_bound(short val, short *list, int groups_size) {
    int u = groups_size;
    int l = 0;
    while (true) {
        if (u == l) {
            return list[l];
        } else if (l + 1 == u) {
            if (list[u] <= val) {
                return list[u];
            } else {
                return list[l];
            }
        }
        int m = (l + u) / 2;
        if (val > list[m]) {
            l = m;
        } else {
            u = m;
        }
    }
}

// returns the smallest value larger than val in list
__device__ short upper_bound(short val, short *list, int groups_size) {
    int u = groups_size;
    int l = 0;
    while (true) {
        if (u == l) {
            return list[l];
        } else if (l + 1 == u) {
            if (list[l] > val) {
                return list[l];
            } else {
                return list[u];
            }
        }
        int m = (l + u) / 2;
        if (val < list[m]) {
            u = m;
        } else {
            l = m;
        }
    }
}

__device__ void
handle_collisions(int i, long communal_cards, long *card_order, short *eval, short *groups, short *coll_vec,
                 int groups_size, float* sorted_range, float* sorted_eval) {
    // Handle collisions before prefix sum consumes sorted_range
    // First two warps handles forward direction
    if (i < 52) {
        int offset = i * 51;
        float sum = 0.0f;
        float group_sum = 0.0f;
        int group_end = upper_bound(coll_vec[offset], groups, groups_size);
        for (int c = 0; c < 51; c++) {
            int index = coll_vec[offset + c];
            // Skip impossible hands, unnecessary here, but consistent
            if ((communal_cards & card_order[eval[index]]) > 0) continue;
            if (index >= group_end) {
                sum += group_sum;
                group_sum = 0.0f;
                group_end = upper_bound(index, groups, groups_size);
            }
            atomicAdd(&sorted_eval[index], -sum);
            group_sum += sorted_range[index];
        }
    }

    // Last two warps handles backwards direction
    if (i >= 64 && i < (52 + 64)) {
        int temp_i = i - 64;
        int offset = temp_i * 51;
        float sum = 0.0f;
        float group_sum = 0.0f;
        int group_end = lower_bound(coll_vec[offset + 50], groups, groups_size);
        for (int c = 0; c < 51; c++) {
            // Go backwards
            int index = coll_vec[offset + 50 - c];
            // Skip impossible hands
            if ((communal_cards & card_order[eval[index]]) > 0) continue;

            if (index < group_end) {
                sum += group_sum;
                group_sum = 0.0f;
                group_end = lower_bound(index, groups, groups_size);
            }
            atomicAdd(&sorted_eval[index], sum);
            group_sum += sorted_range[index];
        }
    }
    __syncthreads();
}

__global__ void
evaluate_showdown_kernel(float *opponent_range, long communal_cards, long *card_order, short *eval, short *groups,
                         short *coll_vec, int groups_size, float *result) {
    // Setup
    __shared__ float sorted_range[1327];
    __shared__ float sorted_eval[1326];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Sort hands by eval
    for (int b = 0; b < 11; b++) {
        int index = i + b * 128;
        if (index < 1326) {
            // reset values
            sorted_range[index] = 0;
            sorted_eval[index] = 0;
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index]]) > 0) continue;
            sorted_range[index] = opponent_range[eval[index]];

        }
    }
    __syncthreads();

    // Handle card collisions
    handle_collisions(i, communal_cards, card_order, eval, groups, coll_vec, groups_size, sorted_range, sorted_eval);

    // Calculate prefix sum
    cuda_prefix_sum(sorted_range);
    if (i == 0) {
        sorted_range[1326] = sorted_range[1325] + opponent_range[eval[1325]];
    }
    __syncthreads();

    // Calculate showdown value of all hands
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index]]) > 0) continue;
            int prev_group = lower_bound(index, groups, groups_size);
            float worse = sorted_range[prev_group];
            int next_group = upper_bound(index, groups, groups_size);
            float better = sorted_range[1326] - sorted_range[next_group];
            sorted_eval[index] += worse - better;
        }
    }

    // Write result
    __syncthreads();
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326) {
            result[eval[index]] = sorted_eval[index];
        }
    }
}


extern "C" {
void prefix_sum_cuda(float *v) {
    size_t input_size = 1326 * sizeof(float);
    float *deviceInput;
    hipMalloc(&deviceInput, input_size);
    hipMemcpy(deviceInput, v, input_size, hipMemcpyHostToDevice);
    gpu_prefix_sum<<<1, 128>>>(deviceInput);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    hipMemcpy(v, deviceInput, input_size, hipMemcpyDeviceToHost);
    hipFree(deviceInput);
}

void evaluate_showdown_cuda(float *opponent_range, long communal_cards, long *card_order, short *eval, short *groups,
                            short *coll_vec, int groups_size, float *result) {
    float *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(float));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(float), hipMemcpyHostToDevice);

    float *device_result;
    hipMalloc(&device_result, 1326 * sizeof(float));

    long *device_card_order;
    hipMalloc(&device_card_order, 1326 * sizeof(long));
    hipMemcpy(device_card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);

    short *device_eval;
    hipMalloc(&device_eval, 1326 * sizeof(short));
    hipMemcpy(device_eval, eval, 1326 * sizeof(short), hipMemcpyHostToDevice);

    short *device_groups;
    hipMalloc(&device_groups, groups_size * sizeof(short));
    hipMemcpy(device_groups, groups, groups_size * sizeof(short), hipMemcpyHostToDevice);

    short *device_coll_vec;
    hipMalloc(&device_coll_vec, 52 * 51 * sizeof(short));
    hipMemcpy(device_coll_vec, coll_vec, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    evaluate_showdown_kernel<<<1, 128>>>(device_opponent_range, communal_cards, device_card_order, device_eval,
                                         device_groups, device_coll_vec, groups_size, device_result);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy(result, device_result, 1326 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_result);
    hipFree(device_card_order);
    hipFree(device_groups);
    hipFree(device_eval);
    hipFree(device_coll_vec);
}
}


/*int main(int argc, char **argv) {
    size_t input_size = 1326*sizeof(float);
    float* input = (float*) malloc(input_size);
    for(int i = 0; i < 1326; i++) {
        input[i] = 1.0;
    }
    prefix_sum(input);
}*/