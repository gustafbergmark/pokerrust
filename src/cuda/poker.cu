#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math.h"
#include "structs.h"
#include "evaluator.cuh"
#include <sys/time.h>
#include <cmath>

__device__ void multiply(Vector *__restrict__ v1, Vector *__restrict__ v2, Vector *__restrict__ res) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            res->values[index] = v1->values[index] * v2->values[index];
        }
    }
}

__device__ void divide(Vector *v1, DataType val) {
    int i = threadIdx.x;
    val = 1.0f/val;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            v1->values[index] *= val;
        }
    }
}

__device__ void fma(Vector *__restrict__ v1, Vector *__restrict__ v2, Vector *__restrict__ res) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            res->values[index] += v1->values[index] * v2->values[index];
        }
    }
}

__device__ void add_assign(Vector *__restrict__ v1, Vector *__restrict__ v2) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            v1->values[index] += v2->values[index];
        }
    }
}

__device__ void sub_assign(Vector *__restrict__ v1, Vector *__restrict__ v2) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            v1->values[index] -= v2->values[index];
        }
    }
}

__device__ void copy(Vector *__restrict__ from, Vector *__restrict__ into) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            into->values[index] = from->values[index];
        }
    }
}


__device__ void zero(Vector *v) {
    int i = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            v->values[index] = 0.0f;
        }
    }
}

__device__ void p_sum(DataType *input, int i) {
    int offset = 1;
    for (int d = TPB / 2; d > 0; d >>= 1) {
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            input[bi] += input[ai];
        }
        offset *= 2;
    }
    if (i == 0) {
        input[TPB - 1] = 0.0f;
    }
    for (int d = 1; d < TPB; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            DataType t = input[ai];
            input[ai] = input[bi];
            input[bi] += t;
        }
    }
    __syncthreads();
}

__device__ void cuda_prefix_sum(DataType *input, DataType *temp) {
    __syncthreads();
    int i = threadIdx.x;
    temp[i] = 0;
    for (int b = 0; b < ITERS; b++) {
        int index = i * ITERS + b;
        if (index < 1326 && i < 127) {
            temp[i] += input[index];
        }
    }
    p_sum(temp, i);

    DataType prefix = temp[i];
    for (int b = 0; b < ITERS; b++) {
        int index = i * ITERS + b;
        if (index < 1326) {
            DataType t = input[index];
            input[index] = prefix;
            prefix += t;
        }
    }
    __syncthreads();
}

__device__ DataType reduce_sum(DataType *vector, DataType *temp) {
    int i = threadIdx.x;
    temp[i] = 0;
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            temp[i] += vector[index];
        }
    }
    __syncthreads();
    for (int k = 64; k > 0; k >>= 1) {
        if (i < k) {
            temp[i] += temp[i + k];
        }
        __syncthreads();
    }
    return temp[0];
}

__device__ void get_strategy(State *state, Vector *scratch, Vector *result) {
    int tid = threadIdx.x;
    Vector *sum = scratch;
    zero(sum);
    int transitions = state->transitions;
    for (int i = 0; i < transitions; i++) {
        add_assign(sum, state->card_strategies[i]);
    }
    for (int i = 0; i < transitions; i++) {
        for (int b = 0; b < ITERS; b++) {
            int index = tid + TPB * b;
            if (index < 1326) {
                if (sum->values[index] <= 1e-4f) {
                    result[i].values[index] = 1.0f / ((DataType) transitions);
                } else {
                    result[i].values[index] = state->card_strategies[i]->values[index] / sum->values[index];
                }
            }
        }
    }
}

__device__ void
get_strategy_abstract(State *state, Vector *scratch, Vector *result, short *abstractions) {
    int tid = threadIdx.x;
    Vector *sum = scratch;
    int transitions = state->transitions;
    DataType vals[2] = {0.0f, 0.0f};
#pragma unroll 3
    for (int k = 0; k < transitions; k++) {
        vals[0] += state->card_strategies[k]->values[tid];
        vals[1] += state->card_strategies[k]->values[tid + 128];
    }
    sum->values[tid] = vals[0];
    sum->values[tid + 128] = vals[1];
    __syncthreads();
    for (int b = 0; b < ITERS; b++) {
        int index = tid + TPB * b;
        if (index < 1326) {
            short abstract_index = abstractions[index];
            for (int k = 0; k < transitions; k++) {
                if (sum->values[abstract_index] <= 1e-4f) {
                    result[k].values[index] = 1.0f / ((DataType) transitions);
                } else {
                    result[k].values[index] =
                            state->card_strategies[k]->values[abstract_index] / sum->values[abstract_index];
                }
            }
        }
    }
    __syncthreads();
}

__device__ void update_strategy(State *__restrict__ state, Vector *__restrict__ update) {
    int tid = threadIdx.x;
    for (int i = 0; i < state->transitions; i++) {
        add_assign(state->card_strategies[i], update + i);
        for (int b = 0; b < ITERS; b++) {
            int index = tid + TPB * b;
            if (index < 1326) {
                state->card_strategies[i]->values[index] = max(state->card_strategies[i]->values[index], 0.0f);
            }
        }
    }
}

__device__ void update_strategy_abstract(State *__restrict__ state, Vector *__restrict__ update,
                                         short *abstractions) {
    int tid = threadIdx.x;
    __syncthreads();
    for (int b = 0; b < ITERS; b++) {
        int index = tid + TPB * b;
        if (index < 1326) {
            short abstract_index = abstractions[index];
            for (int k = 0; k < state->transitions; k++) {
                atomicAdd(&state->card_strategies[k]->values[abstract_index], update[k].values[index]);
            }
        }
    }
    __syncthreads();
    for (int k = 0; k < state->transitions; k++) {
        state->card_strategies[k]->values[tid] = max(state->card_strategies[k]->values[tid], 0.0f);
        state->card_strategies[k]->values[tid + 128] = max(state->card_strategies[k]->values[tid + 128], 0.0f);
    }
    __syncthreads();
}

__device__ void
handle_collisions(short *coll_vec,
                  DataType *sorted_range, DataType *sorted_eval) {
    int i = threadIdx.x;
    __syncthreads();
    // Handle collisions before prefix sum consumes sorted_range
    // First two warps handles forward direction
    if (i < 52) {
        int offset = i * 51;
        DataType sum = 0.0f;
        DataType group_sum = 0.0f;
        for (int c = 0; c < 51; c++) {
            int index = coll_vec[offset + c];
            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0f;
            }
            atomicAdd(&sorted_eval[index & 2047], -sum);
            group_sum += sorted_range[index & 2047];
        }
    }

    // Last two warps handles backwards direction
    if ((i >= 64) && (i < (52 + 64))) {
        int temp_i = i - 64;
        int offset = temp_i * 51;
        DataType sum = 0.0f;
        DataType group_sum = 0.0f;
        for (int c = 0; c < 51; c++) {
            // Go backwards
            int index = coll_vec[offset + 50 - c];
            // Reverse ordering, because reverse iteration
            atomicAdd(&sorted_eval[index & 2047], sum);
            group_sum += sorted_range[index & 2047];

            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0f;
            }
        }
    }
    __syncthreads();
}

__device__ void
evaluate_showdown(DataType *opponent_range, short *eval,
                  short *coll_vec, DataType bet, Vector *scratch,
                  DataType *temp) {
    __syncthreads();
    DataType *result = (DataType *) scratch;
    DataType *sorted_range = (DataType *) (scratch + 1);
    DataType *sorted_eval = (DataType *) (scratch + 2);

    // Setup
    int i = threadIdx.x;
    // Sort hands by eval
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            // reset values
            sorted_eval[index] = 0.0f;
            result[index] = 0.0f;
            sorted_range[index] = opponent_range[eval[index] & 2047];
        }
        if (index == 1326) {
            sorted_range[index] = 0.0f;
        }
    }

    // Handle card collisions
    handle_collisions(coll_vec, sorted_range, sorted_eval);

    // Calculate prefix sum
    cuda_prefix_sum(sorted_range, temp);
    if (i == 0) {
        sorted_range[1326] = sorted_range[1325] + opponent_range[eval[1325] & 2047];
    }
    __syncthreads();

    // Calculate showdown value of all hands
    int prev_group = eval[1326 + i];
    for (int b = 0; b < ITERS; b++) {
        int index = i * ITERS + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if (eval[index] & 2048) { prev_group = index; }
            DataType worse = sorted_range[prev_group];
            sorted_eval[index] += worse;
        }
    }

    int next_group = eval[1326 + 128 + i];
    for (int b = 10; b >= 0; b--) {
        int index = i * ITERS + b;
        if (index < 1326) {
            DataType better = sorted_range[1326] - sorted_range[next_group];
            sorted_eval[index] -= better;
            // Observe reverse order because of reverse iteration
            if (eval[index] & 2048) { next_group = index; }
        }
    }

    // Write result
    __syncthreads();
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            result[eval[index] & 2047] = sorted_eval[index] * bet;
        }
    }
    __syncthreads();
}


__device__ void
evaluate_fold(Vector *opponent_range, short *card_indexes, DataType bet, Vector *result,
              DataType *temp) {
    __syncthreads();
    // Setup
    int i = threadIdx.x;
    copy(opponent_range, result);


    DataType total = reduce_sum(opponent_range->values, temp);

    __syncthreads();
    temp[i] = 0;
    DataType card_sum = 0.0f;
    if (i < 52) {
        for (int c = 0; c < 26; c++) {
            short index = card_indexes[i * 51 + c];
            card_sum += opponent_range->values[index];
        }
        atomicAdd(&temp[i], card_sum);
    } else if ((i >= 64) && (i < (64 + 52))) {
        for (int c = 26; c < 51; c++) {
            short index = card_indexes[(i - 64) * 51 + c];
            card_sum += opponent_range->values[index];
        }
        atomicAdd(&temp[i - 64], card_sum);
    }
    __syncthreads();
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            long cards = from_index(index);
            int card1 = __ffsll(cards) - 1;
            cards -= 1l << card1;
            int card2 = __ffsll(cards) - 1;
            result->values[index] -= temp[card1] + temp[card2];
        }
    }
    __syncthreads();
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            result->values[index] += total;
            result->values[index] *= bet;
        }
    }
}

__device__ void remove_collisions(Vector *vector, long cards) {
    int tid = threadIdx.x;
    for (int b = 0; b < ITERS; b++) {
        int index = tid + TPB * b;
        if (index < 1326) {
            if (from_index(index) & cards) vector->values[index] = 0.0f;
        }
    }
    __syncthreads();
}

__device__ void handle_node(Vector *scratch, Context *contexts, short updating_player, bool calc_exploit, int *depth,
                            long communal_cards, short *abstractions) {
    int tid = threadIdx.x;
    Context *context = &contexts[*depth];
    State *state = context->state;
    int transitions = state->transitions;
    Vector *opponent_range = context->opponent_range;
    Vector *average_strategy = scratch;
    scratch += 1;
    Vector *action_probs = scratch;
    scratch += transitions; // + transitions
    Vector *results = scratch;
    scratch += transitions; // + state-> transitions
    if (context->transition == 0) {
        if ((updating_player == state->next_to_act) && calc_exploit) {
            for (int b = 0; b < ITERS; b++) {
                int index = tid + TPB * b;
                if (index < 1326) {
                    average_strategy->values[index] = -INFINITY;
                }
            }
        } else {
            zero(average_strategy);
        }
        if (__popcll(communal_cards) < 5) {
            get_strategy(state, scratch, action_probs);
        } else {
            get_strategy_abstract(state, scratch, action_probs, abstractions);
        }
    } else {
        int i = context->transition - 1;
        Vector *new_result = average_strategy + 10;
        copy(new_result, results + i);
        if (updating_player == state->next_to_act) {
            if (!calc_exploit) {
                fma(results + i, action_probs + i, average_strategy);
            } else {
                for (int b = 0; b < ITERS; b++) {
                    int index = tid + TPB * b;
                    if (index < 1326) {
                        average_strategy->values[index] = max(average_strategy->values[index],
                                                              (results + i)->values[index]);
                    }
                }
            }
        } else {
            add_assign(average_strategy, results + i);
        }
    }

    if (context->transition == transitions) {
        if ((state->next_to_act == updating_player) && !calc_exploit) {
            for (int i = 0; i < transitions; i++) {
                Vector *util = results + i;
                sub_assign(util, average_strategy);
            }
            if (__popcll(communal_cards) < 5) {
                update_strategy(state, results);
            } else {
                update_strategy_abstract(state, results, abstractions);
            }
        }
        (*depth)--;
    } else {
        int i = context->transition;
        State *next = context->state->next_states[i];
        Vector *new_range;
        if (state->next_to_act == updating_player) {
            new_range = opponent_range;
        } else {
            new_range = scratch;
            scratch += 1; // + 1
            multiply(opponent_range, action_probs + i, new_range);
        }
        contexts[*depth + 1] = {next, new_range, 0};
        context->transition++;
        (*depth)++;
    }
}

__device__ void evaluate_river(Vector *opponent_range_root,
                               State *root_state,
                               Evaluator *evaluator,
                               long communal_cards,
                               short *card_indexes,
                               short *eval,
                               short *coll_vec,
                               Player updating_player,
                               bool calc_exploit,
                               Vector *scratch_root,
                               DataType *temp,
                               short *abstractions) {
    int tid = threadIdx.x;
    __shared__ short local_abstractions[1326];
    for (int b = 0; b < ITERS; b++) {
        int index = tid + 128 * b;
        if (index < 1326) {
            local_abstractions[index] = abstractions[index];
        }
    }
    __syncthreads();
    Context contexts[7];
    contexts[0] = {root_state, opponent_range_root, 0};
    int depth = 0;
    while (depth >= 0) {
        Vector *scratch = scratch_root + depth * 10;
        Context *context = &contexts[depth];
        State *state = context->state;
        Vector *opponent_range = context->opponent_range;

        switch (state->terminal) {
            case Showdown :
                evaluate_showdown(opponent_range->values, eval,
                                  coll_vec, state->sbbet, scratch,
                                  temp);
                depth--;
                break;
            case SBWins :
                evaluate_fold(opponent_range,
                              card_indexes, updating_player == 1 ? -state->bbbet : state->bbbet, scratch,
                              temp);
                depth--;
                break;
            case BBWins :
                evaluate_fold(opponent_range,
                              card_indexes,
                              updating_player == 0 ? -state->sbbet : state->sbbet, scratch, temp);
                depth--;
                break;
            case NonTerminal : {
                handle_node(scratch, contexts, updating_player, calc_exploit, &depth, communal_cards,
                            local_abstractions);
                break;
            }
        }
    }
}

__device__ void evaluate_turn(Vector *opponent_range_root,
                              State *root_state,
                              Evaluator *evaluator,
                              Player updating_player,
                              bool calc_exploit,
                              Vector *scratch_root) {
    __shared__ DataType temp[128];
    // Remove possibility of impossible hands
    remove_collisions(opponent_range_root, root_state->cards);
    Context contexts[7];
    contexts[0] = {root_state, opponent_range_root, 0};
    int depth = 0;

    while (depth >= 0) {
        Vector *scratch = scratch_root + depth * 10;
        Context *context = &contexts[depth];
        State *state = context->state;
        Vector *opponent_range = context->opponent_range;

        switch (state->terminal) {
            case SBWins :
                evaluate_fold(opponent_range,
                              evaluator->card_indexes, updating_player == 1 ? -state->bbbet : state->bbbet, scratch,
                              temp);
                depth--;
                break;
            case BBWins :
                evaluate_fold(opponent_range,
                              evaluator->card_indexes,
                              updating_player == 0 ? -state->sbbet : state->sbbet, scratch, temp);
                depth--;
                break;
            case NonTerminal : {
                handle_node(scratch, contexts, updating_player, calc_exploit, &depth, state->cards, nullptr);
                break;
            }
            case River:
                Vector *result = scratch;
                scratch += 1;
                zero(result);
                State *next = state->next_states[0];
                for (int c = 0; c < 52; c++) {
                    long river = 1l << c;
                    if (river & state->cards) continue;
                    Vector *new_range = scratch;
                    copy(opponent_range, new_range);
                    remove_collisions(new_range, river);
                    long set = (state->cards | river) ^ evaluator->flop;
                    int eval_index = get_index(set);
                    short *eval = evaluator->eval + eval_index * (1326 + 128 * 2);
                    short *coll_vec = evaluator->coll_vec + eval_index * 52 * 51;
                    evaluate_river(new_range, next, evaluator, state->cards | river, evaluator->card_indexes, eval,
                                   coll_vec,
                                   updating_player,
                                   calc_exploit, scratch_root + (depth + 1) * 10, temp,
                                   evaluator->abstractions + eval_index * 1326);
                    remove_collisions(result + 10, river);
                    add_assign(result, result + 10);
                }
                divide(result, 48.0f);
                depth--;
                break;
        }
    }
    // Remove utility of impossible hands
    remove_collisions(scratch_root, root_state->cards);
}

__global__ void evaluate_all(Vector *root_scratch, Vector *opponent_ranges, State *root_states, Evaluator *evaluator,
                             Player updating_player, bool calc_exploit) {
    int block = blockIdx.x;
    Vector *scratch = root_scratch + 10 * 14 * block;
    Vector *opponent_range = opponent_ranges + block / 49;
    State *states = root_states + 270 * block;
    evaluate_turn(opponent_range, states, evaluator, updating_player, calc_exploit, scratch);
}

__global__ void aggregate(Vector *root_scratch, Vector *root_result) {
    int i = threadIdx.x;
    int block = blockIdx.x;
    Vector *scratch = root_scratch + block * 10 * 14 * 49;
    Vector *result = root_result + block;
    for (int i = 1; i < 49; i++) {
        add_assign(result, scratch + 10 * 14 * i);
    }
    for (int b = 0; b < ITERS; b++) {
        int index = i + TPB * b;
        if (index < 1326) {
            result->values[index] /= 49.0f;
        }
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

extern "C" {

void evaluate_cuda(Builder *builder,
                   Evaluator *evaluator,
                   short updating_player,
                   bool calc_exploit) {
    hipError_t err;
    Vector *device_scratch;
    size_t scratch_size = sizeof(Vector) * 10 * 14 * 49 * 63;
    hipMalloc(&device_scratch, scratch_size);
    hipMemset(device_scratch, 0, scratch_size);
    hipMemcpy(builder->opponent_ranges, builder->communication, 63 * sizeof(Vector), hipMemcpyHostToDevice);
    hipMemset(builder->results, 0, 63 * sizeof(Vector));
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Setup error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    evaluate_all<<<49 * 63, 128>>>(device_scratch, builder->opponent_ranges, builder->states, evaluator,
                                   updating_player == 0 ? Small : Big, calc_exploit);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Main execution error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    aggregate<<<63, 128>>>(device_scratch, builder->results);
    hipMemcpy(builder->communication, builder->results, 63 * sizeof(Vector), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Aggregation error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    hipFree(device_scratch);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Execution error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
}
}

#ifdef TEST
#include "builder.cu"
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <thread>

int main() {
    Evaluator *device_evaluator;
    hipMalloc(&device_evaluator, sizeof(Evaluator));
    Evaluator *evaluator = (Evaluator *) calloc(1, sizeof(Evaluator));
    int file_evaluator = open("evaluator_test", O_RDWR | O_CREAT, 0666);
    void *src = mmap(NULL, sizeof(Evaluator), PROT_READ | PROT_WRITE, MAP_SHARED, file_evaluator, 0);

    memcpy(evaluator, src, sizeof(Evaluator));
    munmap(src, sizeof(Evaluator));
    close(file_evaluator);

    hipMemcpy(device_evaluator, evaluator, sizeof(Evaluator), hipMemcpyHostToDevice);
    DataType *range = (float *) calloc(1326, sizeof(DataType));
    for (int i = 0; i < 1326; i++) {
        if (evaluator->card_order[i] & 7l) {
            range[i] = 0.0;
        } else {
            range[i] = 1.0;
        }
    }

    State** states = build_turn_cuda(7l, 1.0);

    DataType *result = (float *) calloc(1326, sizeof(DataType));

    double start = cpuSecond();

    int THREADS = 100;
    std::thread threads[THREADS];
    for(int i = 0; i < THREADS; i++) threads[i] = std::thread(evaluate_turn_cuda, range, states, device_evaluator, 0, true, result);
    for(int i = 0; i < THREADS; i++) threads[i].join();

    double elapsed = cpuSecond() - start;


    float sum = 0;
    for (int i = 0; i < 1326; i++) {
        sum += result[i];
    }
    printf("sum: %f elapsed: %f\n", sum, elapsed);
    free(range);
    free(result);
    for (int i = 0; i < 49; i++) {
        hipFree(states[i]);
    }
    hipFree(device_evaluator);
}
#endif