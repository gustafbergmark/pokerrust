#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math.h"
#include "structs.h"
#include "evaluator.cuh"
#include <sys/time.h>
// Everything expect a  dimension of 1x128, and vectors of size 1326 (most of the time)

__device__ void multiply(Vector* __restrict__ v1, Vector* __restrict__ v2, Vector* __restrict__ res) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            res->values[index] = v1->values[index] * v2->values[index];
        }
    }
}

__device__ void fma(Vector* __restrict__ v1, Vector* __restrict__ v2, Vector* __restrict__ res) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            res->values[index] += v1->values[index] * v2->values[index];
        }
    }
}

__device__ void add_assign(Vector* __restrict__ v1, Vector* __restrict__ v2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v1->values[index] += v2->values[index];
        }
    }
}

__device__ void sub_assign(Vector* __restrict__ v1, Vector* __restrict__ v2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v1->values[index] -= v2->values[index];
        }
    }
}

__device__ void copy(Vector* __restrict__ from, Vector* __restrict__ into) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            into->values[index] = from->values[index];
        }
    }
}


__device__ void zero(Vector *v) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v->values[index] = 0;
        }
    }
}

__device__ void p_sum(DataType *input, int i) {
    int offset = 1;
    for (int d = 64; d > 0; d >>= 1) {
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            input[bi] += input[ai];
        }
        offset *= 2;
    }
    if (i == 0) {
        input[127] = 0;
    }
    for (int d = 1; d < 128; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            DataType t = input[ai];
            input[ai] = input[bi];
            input[bi] += t;
        }
    }
    __syncthreads();
}

__device__ void cuda_prefix_sum(DataType *input, DataType *temp) {
    __syncthreads();
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp[i] = 0;
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326 && i < 127) {
            temp[i] += input[index];
        }
    }
    p_sum(temp, i);

    DataType prefix = temp[i];
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326) {
            DataType t = input[index];
            input[index] = prefix;
            prefix += t;
        }
    }
    __syncthreads();
}

__device__ void get_strategy(State *state, Vector *scratch, Vector *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    Vector *sum = scratch;
    zero(sum);
    for (int i = 0; i < state->transitions; i++) {
        add_assign(sum, state->card_strategies[i]);
    }
    for (int i = 0; i < state->transitions; i++) {
        for (int b = 0; b < 11; b++) {
            int index = tid + 128 * b;
            if (index < 1326) {
                if (sum->values[index] <= 1e-4) {
                    result[i].values[index] = 1.0 / ((DataType) state->transitions);
                } else {
                    result[i].values[index] = state->card_strategies[i]->values[index] / sum->values[index];
                }
            }
        }
    }
}

__device__ void update_strategy(State* __restrict__ state, Vector* __restrict__ update) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < state->transitions; i++) {
        add_assign(state->card_strategies[i], update + i);
        for (int b = 0; b < 11; b++) {
            int index = tid + 128 * b;
            if (index < 1326) {
                state->card_strategies[i]->values[index] = max(state->card_strategies[i]->values[index], 0.0);
            }
        }
    }
}

__device__ void
handle_collisions(int i, long communal_cards, long *card_order, short *eval, short *coll_vec,
                  DataType *sorted_range, DataType *sorted_eval) {
    __syncthreads();
    // Handle collisions before prefix sum consumes sorted_range
    // First two warps handles forward direction
    if (i < 52) {
        int offset = i * 51;
        DataType sum = 0.0;
        DataType group_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            int index = coll_vec[offset + c];
            // Skip impossible hands, unnecessary here, but consistent
            if ((communal_cards & card_order[eval[index & 2047] & 2047]) > 0) continue;
            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0;
            }
            atomicAdd(&sorted_eval[index & 2047], -sum);
            group_sum += sorted_range[index & 2047];
        }
    }

    // Last two warps handles backwards direction
    if ((i >= 64) && (i < (52 + 64))) {
        int temp_i = i - 64;
        int offset = temp_i * 51;
        DataType sum = 0.0;
        DataType group_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            // Go backwards
            int index = coll_vec[offset + 50 - c];
            // Skip impossible hands
            if ((communal_cards & card_order[eval[index & 2047] & 2047]) > 0) continue;
            // Reverse ordering, because reverse iteration
            atomicAdd(&sorted_eval[index & 2047], sum);
            group_sum += sorted_range[index & 2047];

            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0;
            }
        }
    }
    __syncthreads();
}

__device__ void
evaluate_showdown_kernel_inner(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                               short *coll_vec, DataType bet, DataType *result, DataType *sorted_range,
                               DataType *sorted_eval,
                               DataType *temp) {
    __syncthreads();
    // Setup
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Sort hands by eval
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            // reset values
            sorted_range[index] = 0;
            sorted_eval[index] = 0;
            result[index] = 0;
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            sorted_range[index] = opponent_range[eval[index] & 2047];
        }
        if (index == 1326) {
            sorted_range[index] = 0;
        }
    }

    // Handle card collisions
    handle_collisions(i, communal_cards, card_order, eval, coll_vec, sorted_range, sorted_eval);

    // Calculate prefix sum
    cuda_prefix_sum(sorted_range, temp);
    if (i == 0) {
        sorted_range[1326] = sorted_range[1325] + opponent_range[eval[1325] & 2047];
    }
    __syncthreads();

    // Calculate showdown value of all hands
    int prev_group = eval[1326 + i];
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            if (eval[index] & 2048) { prev_group = index; }
            DataType worse = sorted_range[prev_group];
            sorted_eval[index] += worse;
        }
    }

    int next_group = eval[1326 + 128 + i];
    for (int b = 10; b >= 0; b--) {
        int index = i * 11 + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            DataType better = sorted_range[1326] - sorted_range[next_group];
            sorted_eval[index] -= better;
            // Observe reverse order because of reverse iteration
            if (eval[index] & 2048) { next_group = index; }
        }
    }

    // Write result
    __syncthreads();
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            result[eval[index] & 2047] = sorted_eval[index] * bet;
        }
    }
    __syncthreads();
}

__global__ void
evaluate_showdown_kernel(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                         short *coll_vec, DataType bet, DataType *result, Evaluator *evaluator) {
    __shared__ DataType sorted_range[1327];
    __shared__ DataType sorted_eval[1326];
    __shared__ DataType temp[128];
    evaluate_showdown_kernel_inner(opponent_range, communal_cards, card_order, eval, coll_vec, bet, result,
                                   sorted_range, sorted_eval, temp);
}

__device__ void
evaluate_fold_kernel_inner(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                           short updating_player, short folding_player, DataType bet, DataType *result,
                           DataType *range_sum,
                           DataType *temp) {
    __syncthreads();
    // Setup
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            // reset values
            range_sum[index] = 0;
            // Because of inclusion-exclusion, we need to add the
            // probability that the opponent got exactly the same hand
            result[index] = 0;
            // Impossible hand since overlap with communal cards
            if (communal_cards & card_order[index]) continue;
            range_sum[index] = opponent_range[index];
            result[index] = opponent_range[index];
        }
    }

    // Calculate prefix sum
    cuda_prefix_sum(range_sum, temp);

    // using result[1325] is a bit hacky, but correct
    DataType total = range_sum[1325] + result[1325];

    if (i < 52) {
        DataType card_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            short index = card_indexes[i * 51 + c];
            if (communal_cards & card_order[index]) continue;
            card_sum += opponent_range[index];
        }
        for (int c = 0; c < 51; c++) {
            short index = card_indexes[i * 51 + c];
            if (communal_cards & card_order[index]) continue;
            atomicAdd(&result[index], -card_sum);
        }
    }
    __syncthreads();

    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            if (communal_cards & card_order[index]) continue;
            result[index] += total;
            if (updating_player == folding_player) {
                result[index] *= -bet;
            } else {
                result[index] *= bet;
            }
        }
    }
}

__global__ void
evaluate_fold_kernel(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                     short updating_player, short folding_player, DataType bet, DataType *result) {
    __shared__ DataType range_sum[1326];
    __shared__ DataType temp[128];
    evaluate_fold_kernel_inner(opponent_range, communal_cards, card_order, card_indexes, updating_player,
                               folding_player, bet, result, range_sum, temp);
}

__device__ void evaluate_post_turn_kernel_inner(Vector *opponent_range_root,
                                                State *root_state,
                                                Evaluator *evaluator,
                                                Player updating_player,
                                                Vector *scratch_root, DataType *sorted_range, DataType *sorted_eval,
                                                DataType *temp) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    Context contexts[14];
    contexts[0] = {root_state, opponent_range_root, 0};
    int depth = 0;

    while (depth >= 0) {
        //if(tid==0) printf("depth %d\n", depth);
        Vector *scratch = scratch_root + depth * 10;
        Context *context = &contexts[depth];
        State *state = context->state;
        Vector *opponent_range = context->opponent_range;
        switch (state->terminal) {
            case Showdown : {
                long set = state->cards ^ evaluator->flop;
                int eval_index = get_index(set);
                short *eval = evaluator->eval + eval_index * (1326 + 128 * 2);
                short *coll_vec = evaluator->coll_vec + eval_index * 52 * 51;
                evaluate_showdown_kernel_inner(opponent_range->values, state->cards, evaluator->card_order, eval,
                                               coll_vec, state->sbbet, (DataType*)scratch, sorted_range, sorted_eval, temp);
                depth--;
            }
                break;
            case SBWins :
                evaluate_fold_kernel_inner(opponent_range->values, state->cards, evaluator->card_order, evaluator->card_indexes,
                                           updating_player, 1, state->bbbet, (DataType*) scratch, sorted_eval, temp);
                depth--;
                break;
            case BBWins :
                evaluate_fold_kernel_inner(opponent_range->values, state->cards, evaluator->card_order, evaluator->card_indexes,
                                           updating_player, 0, state->sbbet, (DataType*) scratch, sorted_eval, temp);
                depth--;
                break;
            case NonTerminal : {
                Vector *average_strategy = scratch;
                scratch += 1;
                Vector *action_probs = scratch;
                scratch += state->transitions; // + state->transitions
                Vector *results = scratch;
                scratch += state->transitions; // + state-> transitions
                if (context->transition == 0) {
                    zero(average_strategy);
                    get_strategy(state, scratch, action_probs);
                } else {
                    int i = context->transition - 1;
                    Vector* new_result = average_strategy + 10;
                    copy(new_result, results + i);
                    if (updating_player == state->next_to_act) {
                        fma(results + i, action_probs + i, average_strategy);
                    } else {
                        add_assign(average_strategy, results + i);
                    }
                }

                if (context->transition == context->state->transitions) {
                    if (state->next_to_act == updating_player) {
                        for (int i = 0; i < state->transitions; i++) {
                            Vector *util = results + i;
                            sub_assign(util, average_strategy);
                        }
                        update_strategy(state, results);
                    }
                    depth--;
                } else {
                    int i = context->transition;
                    State *next = context->state->next_states[i];
                    Vector *new_range;
                    if (state->next_to_act == updating_player) {
                        new_range = opponent_range;
                    } else {
                        new_range = scratch;
                        scratch += 1; // + 1
                        multiply(opponent_range, action_probs + i, new_range);
                    }
                    contexts[depth + 1] = {next, new_range, 0};
                    context->transition++;
                    depth++;
                }
                break;
            }
            case River:
                Vector *result = scratch;
                scratch += 1;
                if (context->transition == 0) {
                    zero(result);
                } else {
                    // offset to next depths result
                    add_assign(result, result + 10);
                }
                if (context->transition == context->state->transitions) {
                    for (int b = 0; b < 11; b++) {
                        int index = tid + 128 * b;
                        if (index < 1326) {
                            result->values[index] /= ((DataType) state->transitions);
                        }
                    }
                    depth--;
                } else {
                    int i = context->transition;
                    State *next = state->next_states[i];
                    contexts[depth + 1] = {next, opponent_range, 0};
                    context->transition += 1;
                    depth++;
                }
                break;
        }
    }
}

__global__ void evaluate_post_turn_kernel(Vector *opponent_range,
                                          State *state,
                                          Evaluator *evaluator,
                                          Player updating_player,
                                          Vector *scratch) {
    __shared__ DataType sorted_range[1327];
    __shared__ DataType sorted_eval[1326];
    __shared__ DataType temp[128];
    evaluate_post_turn_kernel_inner(opponent_range, state, evaluator, updating_player, scratch, sorted_range,
                                    sorted_eval, temp);
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

extern "C" {
void evaluate_showdown_cuda(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                            short *coll_vec, DataType bet, DataType *result, Evaluator *evaluator) {
    DataType *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(DataType));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);

    DataType *device_result;
    hipMalloc(&device_result, 1326 * sizeof(DataType));

    long *device_card_order;
    hipMalloc(&device_card_order, 1326 * sizeof(long));
    hipMemcpy(device_card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);

    short *device_eval;
    hipMalloc(&device_eval, (1326 + 128 * 2) * sizeof(short));
    hipMemcpy(device_eval, eval, (1326 + 128 * 2) * sizeof(short), hipMemcpyHostToDevice);

    short *device_coll_vec;
    hipMalloc(&device_coll_vec, 52 * 51 * sizeof(short));
    hipMemcpy(device_coll_vec, coll_vec, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    evaluate_showdown_kernel<<<1, 128>>>(device_opponent_range, communal_cards, device_card_order, device_eval,
                                         device_coll_vec, bet, device_result, evaluator);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    fflush(stdout);
    hipMemcpy(result, device_result, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_result);
    hipFree(device_card_order);
    hipFree(device_eval);
    hipFree(device_coll_vec);
}

void evaluate_fold_cuda(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                        short updating_player,
                        short folding_player, DataType bet, DataType *result) {
    DataType *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(DataType));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);

    DataType *device_result;
    hipMalloc(&device_result, 1326 * sizeof(DataType));

    long *device_card_order;
    hipMalloc(&device_card_order, 1326 * sizeof(long));
    hipMemcpy(device_card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);

    short *device_card_indexes;
    hipMalloc(&device_card_indexes, 52 * 51 * sizeof(short));
    hipMemcpy(device_card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    evaluate_fold_kernel<<<1, 128>>>(device_opponent_range, communal_cards, device_card_order, device_card_indexes,
                                     updating_player, folding_player, bet, device_result);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();
    hipMemcpy(result, device_result, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_result);
    hipFree(device_card_order);
    hipFree(device_card_indexes);
}

void evaluate_post_turn_cuda(DataType *opponent_range,
                             State *state,
                             Evaluator *evaluator,
                             short updating_player,
                             DataType *result) {
    Vector *device_opponent_range;
    hipMalloc(&device_opponent_range, sizeof(Vector));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);


    Vector *device_scratch;
    // Max depth less than 14 i think, and max 8 vecs allocated per level
    int scratch_size =  sizeof(Vector) * 10 * 14;
    hipMalloc(&device_scratch, scratch_size);
    hipMemset(device_scratch, 0, scratch_size);
    // Result will always be put in scratch[0..1326]
    for(int i = 0; i < 1; i++ ) {
        double start = cpuSecond();
        evaluate_post_turn_kernel<<<1, 128>>>(device_opponent_range, state, evaluator, updating_player == 0 ? Small : Big,
                                              device_scratch);

        hipMemcpy(result, device_scratch, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(err));
        }
        double elapsed = cpuSecond() - start;
        printf("Kernel time: %fs i: %d\n", elapsed,i);
        fflush(stdout);
    }

    hipFree(device_opponent_range);
    hipFree(device_scratch);
}
}
//
//#include "builder.cu"
//#include <fcntl.h>
//#include <sys/mman.h>
//#include <unistd.h>
//
//int main() {
//    DataType* range = (float*)calloc(1326, sizeof (DataType));
//    for(int i = 0; i < 1326; i++) {
//        range[i] = 1.0;
//    }
//    State* state = build_post_turn_cuda(15l, 1.0);
//    DataType* result = (float*) calloc(1326, sizeof(DataType));
//    Evaluator* device_evaluator;
//    hipMalloc(&device_evaluator, sizeof(Evaluator));
//    Evaluator* evaluator = (Evaluator*) calloc(1, sizeof (Evaluator));
//    int file_evaluator = open("evaluator_test", O_RDWR | O_CREAT, 0666);
//    void *src = mmap(NULL, sizeof(Evaluator), PROT_READ | PROT_WRITE, MAP_SHARED, file_evaluator, 0);
//
//    memcpy(evaluator, src, sizeof (Evaluator));
//    munmap(src, sizeof (Evaluator));
//    close(file_evaluator);
//
//    hipMemcpy(device_evaluator, evaluator, sizeof (Evaluator), hipMemcpyHostToDevice);
//    evaluate_post_turn_cuda(range, state, device_evaluator, 0, result);
//    float sum = 0;
//    for(int i =0 ;i < 1326; i++) {
//        sum += result[i];
//    }
//    printf("sum: %f\n", sum);
//    free(range);
//    free(result);
//    hipFree(state);
//    hipFree(device_evaluator);
//}

//Evaluator* src = (Evaluator*) malloc(sizeof (Evaluator));
//hipMemcpy(src, device_eval, sizeof (Evaluator), hipMemcpyDeviceToHost);
//hipDeviceSynchronize();
///* DESTINATION */
//int dfd = open("evaluator_test", O_RDWR | O_CREAT, 0666);
//size_t filesize = sizeof(Evaluator);
//
//ftruncate(dfd, sizeof (Evaluator));
//
//void* dest = mmap(NULL, sizeof(Evaluator), PROT_READ | PROT_WRITE, MAP_SHARED, dfd, 0);
//
///* COPY */
//
//memcpy(dest, src, filesize);
//
//munmap(dest, filesize);
//close(dfd);
//
//exit(2);