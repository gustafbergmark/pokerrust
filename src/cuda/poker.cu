#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math.h"
#include "structs.h"
#include "evaluator.cuh"
// Everything expect a  dimension of 1x128, and vectors of size 1326 (most of the time)

__device__ void multiply(DataType *v1, DataType *v2, DataType *res) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            res[index] = v1[index] * v2[index];
        }
    }
}

__device__ void fma(DataType *v1, DataType *v2, DataType *res) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            res[index] += v1[index] * v2[index];
        }
    }
}

__device__ void add_assign(DataType *v1, DataType *v2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v1[index] += v2[index];
        }
    }
}

__device__ void sub_assign(DataType *v1, DataType *v2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v1[index] -= v2[index];
        }
    }
}


__device__ void zero(DataType *v) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            v[index] = 0;
        }
    }
}

__device__ void p_sum(DataType *input, int i) {
    int offset = 1;
    for (int d = 64; d > 0; d >>= 1) {
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            input[bi] += input[ai];
        }
        offset *= 2;
    }
    if (i == 0) {
        input[127] = 0;
    }
    for (int d = 1; d < 128; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (i < d) {
            int ai = offset * (2 * i + 1) - 1;
            int bi = offset * (2 * i + 2) - 1;
            DataType t = input[ai];
            input[ai] = input[bi];
            input[bi] += t;
        }
    }
    __syncthreads();
}

__device__ void cuda_prefix_sum(DataType *input, DataType *temp) {
    __syncthreads();
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp[i] = 0;
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326 && i < 127) {
            temp[i] += input[index];
        }
    }
    p_sum(temp, i);

    DataType prefix = temp[i];
    for (int b = 0; b < 11; b++) {
        int index = i * 11+b;
        if (index < 1326) {
            DataType t = input[index];
            input[index] = prefix;
            prefix += t;
        }
    }
    __syncthreads();
}

__device__ void get_strategy(State *state, DataType *scratch, DataType *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    DataType *sum = scratch;
    zero(sum);
    for (int i = 0; i < state->transitions; i++) {
        add_assign(sum, state->card_strategies[i]);
    }
    for (int i = 0; i < state->transitions; i++) {
        for (int b = 0; b < 11; b++) {
            int index = tid + 128* b;
            if (index < 1326) {
                if (sum[index] <= 1e-4) {
                    result[index + i * 1326] = 1.0 / ((DataType) state->transitions);
                } else {
                    result[index + i * 1326] = state->card_strategies[i][index] / sum[index];
                }
            }
        }
    }
}

__device__ void update_strategy(State *state, DataType *update) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < state->transitions; i++) {
        add_assign(state->card_strategies[i], update + 1326 * i);
        for (int b = 0; b < 11; b++) {
            int index = tid + 128 * b;
            if (index < 1326) {
                state->card_strategies[i][index] = max(state->card_strategies[i][index], 0.0);
            }
        }
    }
}

__device__ void
handle_collisions(int i, long communal_cards, long *card_order, short *eval, short *coll_vec,
                  DataType *sorted_range, DataType *sorted_eval) {
    __syncthreads();
    // Handle collisions before prefix sum consumes sorted_range
    // First two warps handles forward direction
    if (i < 52) {
        int offset = i * 51;
        DataType sum = 0.0;
        DataType group_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            int index = coll_vec[offset + c];
            // Skip impossible hands, unnecessary here, but consistent
            if ((communal_cards & card_order[eval[index & 2047] & 2047]) > 0) continue;
            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0;
            }
            atomicAdd(&sorted_eval[index & 2047], -sum);
            group_sum += sorted_range[index & 2047];
        }
    }

    // Last two warps handles backwards direction
    if ((i >= 64) && (i < (52 + 64))) {
        int temp_i = i - 64;
        int offset = temp_i * 51;
        DataType sum = 0.0;
        DataType group_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            // Go backwards
            int index = coll_vec[offset + 50 - c];
            // Skip impossible hands
            if ((communal_cards & card_order[eval[index & 2047] & 2047]) > 0) continue;
            // Reverse ordering, because reverse iteration
            atomicAdd(&sorted_eval[index & 2047], sum);
            group_sum += sorted_range[index & 2047];

            // 2048 bit set => new group
            if (index & 2048) {
                sum += group_sum;
                group_sum = 0.0;
            }
        }
    }
    __syncthreads();
}

__device__ void
evaluate_showdown_kernel_inner(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                               short *coll_vec, DataType bet, DataType *result, DataType *sorted_range,
                               DataType *sorted_eval,
                               DataType *temp) {
    __syncthreads();
    // Setup
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Sort hands by eval
    for (int b = 0; b < 11; b++) {
        int index = i + 128* b;
        if (index < 1326) {
            // reset values
            sorted_range[index] = 0;
            sorted_eval[index] = 0;
            result[index] = 0;
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            sorted_range[index] = opponent_range[eval[index] & 2047];
        }
        if (index == 1326) {
            sorted_range[index] = 0;
        }
    }

    // Handle card collisions
    handle_collisions(i, communal_cards, card_order, eval, coll_vec, sorted_range, sorted_eval);

    // Calculate prefix sum
    cuda_prefix_sum(sorted_range, temp);
    if (i == 0) {
        sorted_range[1326] = sorted_range[1325] + opponent_range[eval[1325] & 2047];
    }
    __syncthreads();

    // Calculate showdown value of all hands
    int prev_group = eval[1326 + i];
    for (int b = 0; b < 11; b++) {
        int index = i * 11 + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            if (eval[index] & 2048) { prev_group = index; }
            DataType worse = sorted_range[prev_group];
            sorted_eval[index] += worse;
        }
    }

    int next_group = eval[1326 + 128 + i];
    for (int b = 10; b >= 0; b--) {
        int index = i * 11 + b;
        if (index < 1326) {
            // Impossible hand since overlap with communal cards
            if ((communal_cards & card_order[eval[index] & 2047]) > 0) continue;
            DataType better = sorted_range[1326] - sorted_range[next_group];
            sorted_eval[index] -= better;
            // Observe reverse order because of reverse iteration
            if (eval[index] & 2048) { next_group = index; }
        }
    }

    // Write result
    __syncthreads();
    for (int b = 0; b < 11; b++) {
        int index = i + 128 * b;
        if (index < 1326) {
            result[eval[index] & 2047] = sorted_eval[index] * bet;
        }
    }
    __syncthreads();
}

__global__ void
evaluate_showdown_kernel(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                         short *coll_vec, DataType bet, DataType *result, Evaluator *evaluator) {
    __shared__ DataType sorted_range[1327];
    __shared__ DataType sorted_eval[1326];
    __shared__ DataType temp[128];
    evaluate_showdown_kernel_inner(opponent_range, communal_cards, card_order, eval, coll_vec, bet, result,
                                   sorted_range, sorted_eval, temp);
}

__device__ void
evaluate_fold_kernel_inner(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                           short updating_player, short folding_player, DataType bet, DataType *result,
                           DataType *range_sum,
                           DataType *temp) {
    __syncthreads();
    // Setup
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int b = 0; b < 11; b++) {
        int index = i + 128* b;
        if (index < 1326) {
            // reset values
            range_sum[index] = 0;
            // Because of inclusion-exclusion, we need to add the
            // probability that the opponent got exactly the same hand
            result[index] = 0;
            // Impossible hand since overlap with communal cards
            if (communal_cards & card_order[index]) continue;
            range_sum[index] = opponent_range[index];
            result[index] = opponent_range[index];
        }
    }

    // Calculate prefix sum
    cuda_prefix_sum(range_sum, temp);

    // using result[1325] is a bit hacky, but correct
    DataType total = range_sum[1325] + result[1325];

    if (i < 52) {
        DataType card_sum = 0.0;
        for (int c = 0; c < 51; c++) {
            short index = card_indexes[i * 51 + c];
            if (communal_cards & card_order[index]) continue;
            card_sum += opponent_range[index];
        }
        for (int c = 0; c < 51; c++) {
            short index = card_indexes[i * 51 + c];
            if (communal_cards & card_order[index]) continue;
            atomicAdd(&result[index], -card_sum);
        }
    }
    __syncthreads();

    for (int b = 0; b < 11; b++) {
        int index = i + 128* b;
        if (index < 1326) {
            if (communal_cards & card_order[index]) continue;
            result[index] += total;
            if (updating_player == folding_player) {
                result[index] *= -bet;
            } else {
                result[index] *= bet;
            }
        }
    }
}

__global__ void
evaluate_fold_kernel(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                     short updating_player, short folding_player, DataType bet, DataType *result) {
    __shared__ DataType range_sum[1326];
    __shared__ DataType temp[128];
    evaluate_fold_kernel_inner(opponent_range, communal_cards, card_order, card_indexes, updating_player,
                               folding_player, bet, result, range_sum, temp);
}

__device__ void evaluate_post_turn_kernel_inner(DataType *opponent_range,
                                                 State *state,
                                                 Evaluator *evaluator,
                                                 Player updating_player,
                                                 DataType *scratch,
                                                 DataType *result, DataType *sorted_range, DataType *sorted_eval,
                                                 DataType *temp) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    switch (state->terminal) {
        case Showdown : {
            long set = state->cards ^ evaluator->flop;
            int eval_index = get_index(set);
            short *eval = evaluator->eval + eval_index * (1326 + 128 * 2);
            short *coll_vec = evaluator->coll_vec + eval_index * 52 * 51;
            evaluate_showdown_kernel_inner(opponent_range, state->cards, evaluator->card_order, eval,
                                           coll_vec, state->sbbet, result, sorted_range, sorted_eval, temp);

        }
            break;
        case SBWins :
            evaluate_fold_kernel_inner(opponent_range, state->cards, evaluator->card_order, evaluator->card_indexes,
                                       updating_player, 1, state->bbbet, result, sorted_eval, temp);
            break;
        case BBWins :
            evaluate_fold_kernel_inner(opponent_range, state->cards, evaluator->card_order, evaluator->card_indexes,
                                       updating_player, 0, state->sbbet, result, sorted_eval, temp);
            break;
        case NonTerminal : {
            DataType *average_strategy = result;
            zero(average_strategy);
            DataType *action_probs = scratch;
            scratch += 1326 * state->transitions; // + state->transitions
            get_strategy(state, scratch, action_probs);
            DataType *results = scratch;
            scratch += 1326 * state->transitions; // + state-> transitions
            for (int i = 0; i < state->transitions; i++) {
                DataType *new_result = results + 1326 * i;
                DataType *action_prob = action_probs + 1326 * i;
                State *next = state->next_states[i];
                DataType *new_range;
                if (state->next_to_act == updating_player) {
                    new_range = opponent_range;
                } else {
                    new_range = scratch;
                    scratch += 1326; // + 1
                    multiply(opponent_range, action_prob, new_range);
                }
                evaluate_post_turn_kernel_inner(new_range, next, evaluator, updating_player, scratch, new_result,
                                                 sorted_range, sorted_eval, temp);
                if (updating_player == state->next_to_act) {
                    fma(new_result, action_prob, average_strategy);
                } else {
                    add_assign(average_strategy, new_result);
                }
            }
            if (state->next_to_act == updating_player) {
                for (int i = 0; i < state->transitions; i++) {
                    DataType *util = results + 1326 * i;
                    sub_assign(util, average_strategy);
                }
                update_strategy(state, results);
            }
        }
            break;
        case River:
            zero(result);
            DataType *new_result = scratch;
            scratch += 1326;
            for (int i = 0; i < state->transitions; i++) {
                zero(new_result);
                State *next = state->next_states[i];
                evaluate_post_turn_kernel_inner(opponent_range, next, evaluator, updating_player, scratch, new_result,
                                                 sorted_range, sorted_eval, temp);
                add_assign(result, new_result);
            }
            for (int b = 0; b < 11; b++) {
                int index = tid + 128 * b;
                if (index < 1326) {
                    result[index] /= ((DataType) state->transitions);
                }
            }
            break;
    }
}

__global__ void evaluate_post_turn_kernel(DataType *opponent_range,
                                      State *state,
                                      Evaluator *evaluator,
                                      Player updating_player,
                                      DataType *scratch,
                                      DataType *result) {
    __shared__ DataType sorted_range[1327];
    __shared__ DataType sorted_eval[1326];
    __shared__ DataType temp[128];
    evaluate_post_turn_kernel_inner(opponent_range, state, evaluator, updating_player, scratch, result, sorted_range,
                                     sorted_eval, temp);
}


extern "C" {
void evaluate_showdown_cuda(DataType *opponent_range, long communal_cards, long *card_order, short *eval,
                            short *coll_vec, DataType bet, DataType *result, Evaluator *evaluator) {
    DataType *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(DataType));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);

    DataType *device_result;
    hipMalloc(&device_result, 1326 * sizeof(DataType));

    long *device_card_order;
    hipMalloc(&device_card_order, 1326 * sizeof(long));
    hipMemcpy(device_card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);

    short *device_eval;
    hipMalloc(&device_eval, (1326 + 128 * 2) * sizeof(short));
    hipMemcpy(device_eval, eval, (1326 + 128 * 2) * sizeof(short), hipMemcpyHostToDevice);

    short *device_coll_vec;
    hipMalloc(&device_coll_vec, 52 * 51 * sizeof(short));
    hipMemcpy(device_coll_vec, coll_vec, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    evaluate_showdown_kernel<<<1, 128>>>(device_opponent_range, communal_cards, device_card_order, device_eval,
                                         device_coll_vec, bet, device_result, evaluator);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    fflush(stdout);
    hipMemcpy(result, device_result, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_result);
    hipFree(device_card_order);
    hipFree(device_eval);
    hipFree(device_coll_vec);
}

void evaluate_fold_cuda(DataType *opponent_range, long communal_cards, long *card_order, short *card_indexes,
                        short updating_player,
                        short folding_player, DataType bet, DataType *result) {
    DataType *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(DataType));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);

    DataType *device_result;
    hipMalloc(&device_result, 1326 * sizeof(DataType));

    long *device_card_order;
    hipMalloc(&device_card_order, 1326 * sizeof(long));
    hipMemcpy(device_card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);

    short *device_card_indexes;
    hipMalloc(&device_card_indexes, 52 * 51 * sizeof(short));
    hipMemcpy(device_card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    evaluate_fold_kernel<<<1, 128>>>(device_opponent_range, communal_cards, device_card_order, device_card_indexes,
                                     updating_player, folding_player, bet, device_result);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();
    hipMemcpy(result, device_result, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_result);
    hipFree(device_card_order);
    hipFree(device_card_indexes);
}

void evaluate_post_turn_cuda(DataType *opponent_range,
                         State *state,
                         Evaluator *evaluator,
                         short updating_player,
                         DataType *result) {
    DataType *device_opponent_range;
    hipMalloc(&device_opponent_range, 1326 * sizeof(DataType));
    hipMemcpy(device_opponent_range, opponent_range, 1326 * sizeof(DataType), hipMemcpyHostToDevice);

    DataType *device_result;
    hipMalloc(&device_result, 1326 * sizeof(DataType));

    DataType *device_scratch;
    int scratch_size = 1326 * sizeof(DataType) * (7 *
                                                  6 +
                                                  10); // Max 7 ( 3 results, 3 action probs, 1 new_probs) vectors per level, max depth of 6 and 1 for river
    hipMalloc(&device_scratch, scratch_size);
    hipMemset(device_scratch, 0, scratch_size);

    evaluate_post_turn_kernel<<<1, 128>>>(device_opponent_range, state, evaluator, updating_player == 0 ? Small : Big,
                                      device_scratch,
                                      device_result);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    fflush(stdout);
    hipMemcpy(result, device_result, 1326 * sizeof(DataType), hipMemcpyDeviceToHost);

    hipFree(device_opponent_range);
    hipFree(device_scratch);
    hipFree(device_result);
}
}
