#include "hip/hip_runtime.h"
//
// Created by gustaf on 2024-01-22.
//
#include "evaluator.cuh"
#include <stdio.h>
#include <bit>
#include <hip/hip_runtime.h>



__device__ int choose2(int n) {
    if (n>1) {
        return (n*(n-1)) / 2;
    } else {
        return 0;
    }
}

__device__ int get_index(long set) {
    int res = 0;
    int i = __ffsll(set)-1;
    res += i;
    set ^= 1l << i;
    i = __ffsll(set)-1;
    res += choose2(i);
    return res;
}

__device__ long from_index(int index) {
    int limit = (int)(sqrtf((float)(2*index))) + 1;
    //if (limit*(limit-1)/2 > index) limit--;
    limit -= (limit*(limit-1)) > (2*index);
    index -= limit * (limit-1) / 2;
    return (1l<<(limit)) | ( 1l << index);
}



extern "C" {
Evaluator *transfer_flop_eval_cuda(long flop, long *card_order, short *card_indexes, short *eval, short *coll_vec) {
    hipError_t err;
    Evaluator *device_eval;
    hipMalloc(&device_eval, sizeof(Evaluator));
    hipMemcpy(&device_eval->flop, &flop, sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->eval, eval, 1326 * (1326 + 128 * 2) * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->coll_vec, coll_vec, 1326 * 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    return device_eval;
}
void free_eval_cuda(Evaluator *device_eval) {
    hipFree(device_eval);
}

}
//#include <fcntl.h>
//#include <sys/mman.h>
//#include <unistd.h>
//Evaluator* src = (Evaluator*) malloc(sizeof (Evaluator));
//hipMemcpy(src, device_eval, sizeof (Evaluator), hipMemcpyDeviceToHost);
//hipDeviceSynchronize();
///* DESTINATION */
//int dfd = open("evaluator_test", O_RDWR | O_CREAT, 0666);
//size_t filesize = sizeof(Evaluator);
//
//ftruncate(dfd, sizeof (Evaluator));
//
//void* dest = mmap(NULL, sizeof(Evaluator), PROT_READ | PROT_WRITE, MAP_SHARED, dfd, 0);
//
///* COPY */
//
//memcpy(dest, src, filesize);
//
//munmap(dest, filesize);
//close(dfd);
//
//exit(2);