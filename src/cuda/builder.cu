#include <stdio.h>
#include <bit>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "evaluator.cuh"


int possible_actions(State *state, short raises, Action *result) {
    switch (state->action) {
        case Fold :
            return 0;
        case Check :
            result[0] = Call;
            result[1] = Raise;
            return 2;
        case Call :
            if (__builtin_popcountll(state->cards) == 4) {
                result[0] = DealRiver;
                return 1;
            } else {
                return 0;
            }
        case Raise :
            result[0] = Fold;
            result[1] = Call;
            if (raises < 4) {
                result[2] = Raise;
                return 3;
            } else {
                return 2;
            }
        case DealRiver :
            result[0] = Check;
            result[1] = Raise;
            return 2;
    }
    return 0;
}


int get_action(State *state, Action action, State *new_states) {
    Player opponent = state->next_to_act == Small ? Big : Small;
    TerminalState fold_winner = state->next_to_act == Small ? BBWins : SBWins;
    DataType other_bet = state->next_to_act == Small ? state->bbbet : state->sbbet;

    // Copy state and reset some values
    *new_states = *state;
    new_states->next_to_act = opponent;
    new_states->action = action;
    new_states->transitions = 0;

    switch (action) {
        case Fold :
            new_states->terminal = fold_winner;
            break;
        case Check:
            new_states->terminal = NonTerminal;
            break;
        case Call:
            new_states->terminal = __builtin_popcountll(state->cards) == 4 ? River : Showdown;
            new_states->sbbet = other_bet;
            new_states->bbbet = other_bet;
            break;
        case Raise :
            new_states->terminal = NonTerminal;
            new_states->sbbet = state->next_to_act == Small ? state->bbbet + 2.0 : state->sbbet;
            new_states->bbbet = state->next_to_act == Big ? state->sbbet + 2.0 : state->bbbet;
            break;
        case DealRiver:
            for (int c = 0; c < 52; c++) {
                long card = 1l << c;
                if (card & state->cards) continue;
                *new_states = *state;
                new_states->next_to_act = Small;
                new_states->action = action;
                new_states->transitions = 0;
                new_states->cards |= card;
                new_states->terminal = NonTerminal;
                break;
            }
    }
    return 1;
}

void
add_transition(State *parent, State *child, State *root, State *device_root,
               AbstractVector *abstract_vectors, int *abstract_vector_index) {
    if (parent->terminal == NonTerminal) {
        parent->card_strategies[parent->transitions] = (abstract_vectors + *abstract_vector_index);
        *abstract_vector_index += 1;
    }
    // Update pointers to work on gpu;
    parent->next_states[parent->transitions] = device_root + (child - root);
    child->parent = device_root + (parent - root);
    parent->transitions += 1;
}


int build(State *state, short raises, State *root, State *device_root, int *state_index,
          AbstractVector *abstract_vectors, int *abstract_vector_index) {
    Action actions[3] = {};
    int count = 1;
    int num_actions = possible_actions(state, raises, actions);
    for (int i = 0; i < num_actions; i++) {
        Action action = actions[i];
        int new_raises = action == Raise ? raises + 1 : 0;

        State *new_state = root + *state_index;
        get_action(state, action, new_state);
        *state_index += 1;
        count += build(new_state, new_raises, root, device_root, state_index,
                       abstract_vectors, abstract_vector_index);
        add_transition(state, new_state, root, device_root, abstract_vectors,
                       abstract_vector_index);


    }
    return count;
}

void
build_river(long cards, DataType bet, State *root, State *device_root, int *state_index,
            AbstractVector *abstract_vectors, int *abstract_vector_index) {
    *root = {.terminal = River,
            .action = Call,
            .cards = cards,
            .sbbet = bet,
            .bbbet = bet,
            .next_to_act = Small,
            .transitions = 0,
            .card_strategies = {},
            .next_states =  {}};
    *state_index += 1;
    int count = build(root, 0, root, device_root, state_index, abstract_vectors,
                      abstract_vector_index);
//    printf("count: %d\n",count); // 28
//    fflush(stdout);
}
