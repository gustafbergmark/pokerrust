#include <stdio.h>
#include <bit>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "evaluator.cuh"


int possible_actions(State *state, short raises, Action *result) {
    switch (state->action) {
        case Fold :
            return 0;
        case Check :
            result[0] = Call;
            result[1] = Raise;
            return 2;
        case Call :
            if (__builtin_popcountll(state->cards) == 4) {
                result[0] = DealRiver;
                return 1;
            } else {
                return 0;
            }
        case Raise :
            result[0] = Fold;
            result[1] = Call;
            if (raises < 4) {
                result[2] = Raise;
                return 3;
            } else {
                return 2;
            }
        case DealRiver :
            result[0] = Check;
            result[1] = Raise;
            return 2;
        case DealTurn :
            result[0] = Check;
            result[1] = Raise;
            return 2;
    }
}


int get_action(State *state, Action action, State *new_states) {
    Player opponent = state->next_to_act == Small ? Big : Small;
    TerminalState fold_winner = state->next_to_act == Small ? BBWins : SBWins;
    DataType other_bet = state->next_to_act == Small ? state->bbbet : state->sbbet;

    // Copy state and reset some values
    *new_states = *state;
    new_states->next_to_act = opponent;
    new_states->action = action;
    new_states->transitions = 0;

    switch (action) {
        case Fold :
            new_states->terminal = fold_winner;
            break;
        case Check:
            new_states->terminal = NonTerminal;
            break;
        case Call:
            new_states->terminal = __builtin_popcountll(state->cards) == 4 ? River : Showdown;
            new_states->sbbet = other_bet;
            new_states->bbbet = other_bet;
            break;
        case Raise :
            new_states->terminal = NonTerminal;
            new_states->sbbet = state->next_to_act == Small ? state->bbbet + 2.0 : state->sbbet;
            new_states->bbbet = state->next_to_act == Big ? state->sbbet + 2.0 : state->bbbet;
            break;
        case DealRiver:
            for (int c = 0; c < 52; c++) {
                long card = 1l << c;
                if (card & state->cards) continue;
                *new_states = *state;
                new_states->next_to_act = Small;
                new_states->action = action;
                new_states->transitions = 0;
                new_states->cards |= card;
                new_states->terminal = NonTerminal;
                new_states += 1;
            }
            return 48;
    }
    return 1;
}

void
add_transition(State *parent, State *child, DataType *vectors, int *vector_index, State *root, State *device_root) {
    if (parent->terminal == NonTerminal) {
        parent->card_strategies[parent->transitions] = vectors + (*vector_index * 1326);
        *vector_index += 1;
    }
    // Update pointers to work on gpu;
    parent->next_states[parent->transitions] = device_root + (child - root);
    parent->transitions += 1;
}


int build(State *state, short raises, State *root, State *device_root, DataType *vectors, int *state_index,
          int *vector_index) {
    Action actions[3] = {};
    int count = 1;
    int num_actions = possible_actions(state, raises, actions);
    for (int i = 0; i < num_actions; i++) {
        Action action = actions[i];
        int new_raises = action == Raise ? raises + 1 : 0;
        State *new_states = root + *state_index;
        int num_states = get_action(state, action, new_states);
        *state_index += num_states;
        for (int j = 0; j < num_states; j++) {
            State *new_state = new_states + j;
            count += build(new_state, new_raises, root, device_root, vectors, state_index, vector_index);
            add_transition(state, new_state, vectors, vector_index, root, device_root);
        }
    }
    return count;
}

void
build_post_turn_kernel(long cards, DataType bet, State *root, State *device_root, DataType *vectors, int *state_index,
                       int *vector_index) {
    *root = {.terminal = NonTerminal,
            .action = DealTurn,
            .cards = cards,
            .sbbet = bet,
            .bbbet = bet,
            .next_to_act = Small,
            .transitions = 0,
            .card_strategies = {},
            .next_states =  {}};
    *state_index += 1;
    build(root, 0, root, device_root, vectors, state_index, vector_index);
}


extern "C" {
void init() {
    size_t *size = (size_t *) malloc(sizeof(size_t));
    hipDeviceGetLimit(size, hipLimitStackSize);
    printf("old stack size: %zu\n", *size);
    // Allocate 8 GiBi heap
    size_t heap_size = 8l * 1024l * 1024l * 1024l;
    hipDeviceSetLimit(hipLimitStackSize, 2 * 4096);
    hipDeviceGetLimit(size, hipLimitStackSize);
    printf("new stack size: %zu\n", *size);
    fflush(stdout);
}
State *build_post_turn_cuda(long cards, DataType bet) {
    hipError_t err;
    int vector_index = 0;
    int state_index = 0;
    int state_size = sizeof(State) * (27 * 48 * 9 + 27);
    State *root = (State *) malloc(state_size);

    State *device_root;
    hipMalloc(&device_root, state_size);

    DataType *vectors;
    int vectors_size = sizeof(DataType) * 1326 * (26 * 48 * 9 + 26);
    hipMalloc(&vectors, vectors_size);
    hipMemset(vectors, 0, vectors_size);

    build_post_turn_kernel(cards, bet, root, device_root, vectors, &state_index, &vector_index);
    hipMemcpy(device_root, root, state_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Build error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    return device_root;
}
}
