#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "structs.h"


__device__ int possible_actions(State *state, short raises, Action *result) {
    switch (state->action) {
        case Fold :
            return 0;
        case Check :
            result[0] = Call;
            result[1] = Raise;
            return 2;
        case Call :
            if (__popcll(state->cards) == 4) {
                result[0] = DealRiver;
                return 1;
            } else {
                return 0;
            }
        case Raise :
            result[0] = Fold;
            result[1] = Call;
            if (raises < 4) {
                result[2] = Raise;
                return 3;
            } else {
                return 2;
            }
        case DealRiver :
            result[0] = Check;
            result[1] = Raise;
            return 2;
    }
}


__device__ int get_action(State *state, Action action, State *new_states) {
    Player opponent = state->next_to_act == Small ? Big : Small;
    TerminalState fold_winner = state->next_to_act == Small ? BBWins : SBWins;
    DataType other_bet = state->next_to_act == Small ? state->bbbet : state->sbbet;

    // Copy state and reset some values
    *new_states = *state;
    new_states->next_to_act = opponent;
    new_states->action = action;
    new_states->transitions = 0;

    switch (action) {
        case Fold :
            new_states->terminal = fold_winner;
            break;
        case Check:
            new_states->terminal = NonTerminal;
            break;
        case Call:
            new_states->terminal = __popcll(state->cards) == 4 ? River : Showdown;
            new_states->sbbet = other_bet;
            new_states->bbbet = other_bet;
            break;
        case Raise :
            new_states->terminal = NonTerminal;
            new_states->sbbet = state->next_to_act == Small ? state->bbbet + 2.0 : state->sbbet;
            new_states->bbbet = state->next_to_act == Big ? state->sbbet + 2.0 : state->bbbet;
            break;
        case DealRiver:
            for (int c = 0; c < 52; c++) {
                long card = 1l << c;
                if (card & state->cards) continue;
                *new_states = *state;
                new_states->next_to_act = Small;
                new_states->action = action;
                new_states->transitions = 0;
                new_states->cards |= card;
                new_states->terminal = NonTerminal;
                new_states += 1;
            }
            return 48;
    }
    return 1;
}

__device__ void add_transition(State *parent, State *child, DataType *vectors, int *vector_index) {
    if (parent->terminal == NonTerminal) {
        parent->card_strategies[parent->transitions] = vectors + (*vector_index * 1326);
        *vector_index += 1;
    }
    parent->next_states[parent->transitions] = child;
    parent->transitions += 1;
}


__device__ int build(State *state, short raises, State *root, DataType *vectors, int *state_index, int *vector_index) {
    Action actions[3] = {};
    int count = 1;
    int num_actions = possible_actions(state, raises, actions);
    for (int i = 0; i < num_actions; i++) {
        Action action = actions[i];
        int new_raises = action == Raise ? raises + 1 : 0;
        State *new_states = root + *state_index;
        int num_states = get_action(state, action, new_states);
        *state_index += num_states;
        for(int j = 0; j < num_states; j++) {
            State* new_state = new_states + j;
            count += build(new_state, new_raises, root, vectors, state_index, vector_index);
            add_transition(state, new_state, vectors, vector_index);
        }
    }
    return count;
}

__global__ void
build_river_kernel(long cards, DataType bet, State *root, DataType *vectors, int *state_index, int *vector_index) {
    *root = {.terminal = River,
            .action = Call,
            .cards = cards,
            .sbbet = bet,
            .bbbet = bet,
            .next_to_act = Small,
            .transitions = 0,
            .card_strategies = {},
            .next_states =  {}};
    *state_index += 1;
    build(root, 0, root, vectors, state_index, vector_index);
}


extern "C" {
void init() {
    size_t *size = (size_t *) malloc(sizeof(size_t));
    hipDeviceGetLimit(size, hipLimitMallocHeapSize);
    printf("old heap size: %zu\n", *size);
    hipDeviceGetLimit(size, hipLimitStackSize);
    printf("old stack size: %zu\n", *size);
    // Allocate 6 GiBi heap
    size_t heap_size = 8l * 1024l * 1024l * 1024l;
    hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size);
    // Too small stack will result in: unspecified launch failure
    hipDeviceSetLimit(hipLimitStackSize, 2 * 4096);
    hipDeviceGetLimit(size, hipLimitMallocHeapSize);
    printf("new heap size: %zu\n", *size);
    hipDeviceGetLimit(size, hipLimitStackSize);
    printf("new stack size: %zu\n", *size);
    fflush(stdout);
}
State *build_river_cuda(long cards, DataType bet) {
    hipError_t err;
    int vector_index = 0;
    int state_index = 0;
    int *device_state_index;
    hipMalloc(&device_state_index, sizeof(int));
    hipMemcpy(device_state_index, &state_index, sizeof(int), hipMemcpyHostToDevice);

    int *device_vector_index;
    hipMalloc(&device_vector_index, sizeof(int));
    hipMemcpy(device_vector_index, &vector_index, sizeof(int), hipMemcpyHostToDevice);

    State *root;
    hipMalloc(&root, sizeof(State) * (27 * 48 + 1));

    DataType *vectors;
    int vectors_size = sizeof(DataType) * 26 * 1326 * 48;
    hipMalloc(&vectors, vectors_size);
    hipMemset(vectors, 0, vectors_size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("pre Error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }

    build_river_kernel<<<1, 1>>>(cards, bet, root, vectors, device_state_index, device_vector_index);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }

    hipMemcpy(&state_index, device_state_index, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&vector_index, device_vector_index, sizeof(int), hipMemcpyDeviceToHost);
    printf("pointer: %p state_index: %d vector_index: %d\n", root, state_index, vector_index);
    fflush(stdout);

    hipFree(device_state_index);
    hipFree(device_vector_index);
    return root;
}
Evaluator *transfer_post_river_eval_cuda(long *card_order, short *card_indexes, short *eval, short *coll_vec) {
    hipError_t err;
    Evaluator *device_eval;
    hipMalloc(&device_eval, sizeof(Evaluator));
    hipMemcpy(&device_eval->card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->eval, eval, (1326 + 128 * 2) * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->coll_vec, coll_vec, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }

    return device_eval;
}
void free_eval_cuda(Evaluator *device_eval) {
    hipFree(device_eval);
}
}
