#include "hip/hip_runtime.h"
//
// Created by gustaf on 2024-02-09.
//
#include "poker.cu"
#include "structs.h"
#include "evaluator.cuh"
#include "builder.cu"
#include <fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <filesystem>
#include <iostream>


extern "C" {

void evaluate_cuda(Builder *builder,
                   Evaluator *evaluator,
                   short updating_player,
                   bool calc_exploit) {
    hipError_t err;
    Vector *device_scratch;
    size_t scratch_size = sizeof(Vector) * 63 * TURNS * 9 * 10; // 10 scratch per kernel
    hipMalloc(&device_scratch, scratch_size);
    hipMemset(device_scratch, 0, scratch_size);
    hipMemcpy(builder->opponent_ranges, builder->communication, 63 * TURNS * 9 * sizeof(Vector),
               hipMemcpyHostToDevice);
    hipMemset(builder->results, 0, 63 * TURNS * 9 * sizeof(Vector));
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Setup error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
//    for(int i = 0; i < 1326; i++) {
//        printf("%d %f\n", i, builder->communication[0].values[i]);
//    }
//    fflush(stdout);
    evaluate_all<<< 63 * TURNS * 9, TPB>>>(builder->opponent_ranges, builder->results, builder->device_states,
                                           evaluator,
                                           updating_player == 0 ? Small : Big, calc_exploit, device_scratch);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Main execution error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    apply_updates<<<63 * 9, TPB>>>(builder->device_states, updating_player == 0 ? Small : Big);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Apply update error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    hipMemcpy(builder->communication, builder->results, 63 * TURNS * 9 * sizeof(Vector), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_scratch);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Aggregation error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    fflush(stdout);

}
Evaluator *transfer_flop_eval_cuda(long flop, long *card_order, short *card_indexes, short *eval, short *coll_vec,
                                   short *abstractions, long *turns, long *rivers) {
    hipError_t err;
    Evaluator *device_eval;
    hipMalloc(&device_eval, sizeof(Evaluator));
    hipMemcpy(&device_eval->flop, &flop, sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->turns, &turns, TURNS * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->rivers, &rivers, TURNS * RIVERS * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->eval, eval, 1326 * (1326 + 256 * 2) * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->coll_vec, coll_vec, 1326 * 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->abstractions, abstractions, 1326 * 1326 * sizeof(short), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Evaluator error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    return device_eval;
}
void free_eval_cuda(Evaluator *device_eval) {
    hipFree(device_eval);
}
Builder *init_builder() {
    Builder *builder = (Builder *) calloc(1, sizeof(Builder));
    builder->current_index = 0;
    hipMalloc(&builder->device_states, 63 * 9 * 28 * sizeof(State));
    //builder->memory_abstract_vectors = (AbstractVector *) calloc(63 * 9 * 26 * 1755, sizeof(AbstractVector));
    //if (builder->memory_abstract_vectors == NULL) printf("Failed to allocate blob memory\n");
    hipMalloc(&builder->abstract_vectors, 63 * 9 * 26 * sizeof(AbstractVector));
    hipMalloc(&builder->updates, 63 * 9 * 26 * sizeof(AbstractVector));
    hipMemset(builder->abstract_vectors, 0, 63 * 26 * 9 * sizeof(AbstractVector));
    hipHostMalloc(&builder->communication, 63 * TURNS * 9 * sizeof(Vector));
    hipMalloc(&builder->opponent_ranges, 63 * TURNS * 9 * sizeof(Vector));
    hipMalloc(&builder->results, 63 * TURNS * 9 * sizeof(Vector));
    printf("GPU builder created\n");
    fflush(stdout);
    return builder;
}

//void set_memory_c(Builder *builder, AbstractVector *memory) {
//    builder->memory_abstract_vectors = memory;
//}

//void load_blob(Builder *builder) {
//    int file = open("./files/blob.bin", O_RDWR, 0600);
//    void *src = mmap(NULL, sizeof(float) * 63 * 9 * 26 * 1755 * 256,
//                     PROT_READ | PROT_WRITE, MAP_SHARED, file, 0);
//    if (src == MAP_FAILED) {
//        printf("mmap failed");
//    }
//    memcpy(builder->memory_abstract_vectors, src, sizeof(float) * 63 * 9 * 26 * 1755 * 256);
//    fflush(stdout);
//    munmap(src, sizeof(float) * 63 * 9 * 26 * 1755 * 256);
//    close(file);
//}
//
//void save_blob(Builder *builder) {
//    float hash = 0;
//    for (int i = 0; i < 63 * 9 * 26 * 1755; i++) {
//        for (int j = 0; j < ABSTRACTIONS; j++) {
//            float val = builder->memory_abstract_vectors[i].values[j];
//            hash *= fmod(val, 1000);
//            hash = fmod(hash, 1000);
//        }
//    }
//    printf("Saved GPU blob with hash %f\n", hash);
//    int file = open("./files/blob.bin", O_RDWR, 0600);
//    void *src = mmap(NULL, sizeof(float) * 63 * 9 * 26 * 1755 * 256,
//                     PROT_READ | PROT_WRITE, MAP_SHARED, file, 0);
//    if (src == MAP_FAILED) {
//        printf("mmap failed");
//    }
//    memcpy(src, builder->memory_abstract_vectors, sizeof(float) * 63 * 9 * 26 * 1755 * 256);
//    fflush(stdout);
//    munmap(src, sizeof(float) * 63 * 9 * 26 * 1755 * 256);
//    close(file);
//}

void upload_c(Builder *builder, int index, DataType *vector) {
    memcpy(builder->communication + index, vector, 1326 * sizeof(DataType));
}

void download_c(Builder *builder, int index, DataType *vector) {
    memcpy(vector, builder->communication + index, 1326 * sizeof(DataType));
}

void upload_strategy_c(Builder *builder, DataType *source) {
    hipMemcpy(builder->abstract_vectors, source,
               63 * 9 * 26 * sizeof(AbstractVector), hipMemcpyHostToDevice);
}

void download_strategy_c(Builder *builder, DataType *dest) {
    hipMemcpy(dest, builder->abstract_vectors,
               63 * 9 * 26 * sizeof(AbstractVector), hipMemcpyDeviceToHost);
}

int build_river_cuda(long cards, DataType bet, Builder *builder) {
    hipError_t err;
    int start = builder->current_index;
    int abstract_vector_index = 0;
    int state_index = 0;
    int state_size = sizeof(State) * (28);
    State *root = (State *) malloc(state_size);

    State *device_root = builder->device_states + (builder->current_index % 567) * 28;
    AbstractVector *abstract_vectors = builder->abstract_vectors + (builder->current_index % 567) * 26;
    AbstractVector *updates = builder->updates + (builder->current_index % 567) * 26;
    builder->current_index += 1;

    build_river(cards, bet, root, device_root, &state_index, abstract_vectors, updates,
                &abstract_vector_index);
    hipMemcpy(device_root, root, state_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Build error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
//    printf("index: %d\n", start); // 567
//    printf("vector index: %d\n", abstract_vector_index);
//    fflush(stdout);
    free(root);
    return start % 567;
}
}