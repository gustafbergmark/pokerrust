//
// Created by gustaf on 2024-02-09.
//
#include "poker.cu"
#include "structs.h"
#include "evaluator.cuh"
#include "builder.cu"

extern "C" {

void evaluate_cuda(Builder *builder,
                   Evaluator *evaluator,
                   short updating_player,
                   bool calc_exploit) {
    hipError_t err;
    Vector *device_scratch;
    size_t scratch_size = sizeof(Vector) * 63 * 49 * 9 * 10; // 10 scratch per kernel
    hipMalloc(&device_scratch, scratch_size);
    hipMemset(device_scratch, 0, scratch_size);
    hipMemcpy(builder->opponent_ranges, builder->communication, 63 * 49 * 9 * sizeof(Vector), hipMemcpyHostToDevice);
    hipMemset(builder->results, 0, 63 * 49 * 9 * sizeof(Vector));
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Setup error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
//    for(int i = 0; i < 1326; i++) {
//        printf("%d %f\n", i, builder->communication[0].values[i]);
//    }
//    fflush(stdout);
    evaluate_all<<< 63 * 49 * 9, TPB>>>(builder->opponent_ranges, builder->results, builder->states, evaluator,
                                        updating_player == 0 ? Small : Big, calc_exploit, device_scratch);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Main execution error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    hipMemcpy(builder->communication, builder->results, 63 * 49 * 9 * sizeof(Vector), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(device_scratch);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Aggregation error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    fflush(stdout);

}
Evaluator *transfer_flop_eval_cuda(long flop, long *card_order, short *card_indexes, short *eval, short *coll_vec,
                                   short *abstractions) {
    hipError_t err;
    Evaluator *device_eval;
    hipMalloc(&device_eval, sizeof(Evaluator));
    hipMemcpy(&device_eval->flop, &flop, sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_order, card_order, 1326 * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->card_indexes, card_indexes, 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->eval, eval, 1326 * (1326 + 256 * 2) * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->coll_vec, coll_vec, 1326 * 52 * 51 * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(&device_eval->abstractions, abstractions, 1326 * 1326 * sizeof(short), hipMemcpyHostToDevice);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Evaluator error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
    return device_eval;
}
void free_eval_cuda(Evaluator *device_eval) {
    hipFree(device_eval);
}
Builder *init_builder() {
    Builder *builder = (Builder *) calloc(1, sizeof(Builder));
    builder->current_index = 0;
    hipMalloc(&builder->states, 63 * 49 * 9 * 28 * sizeof(State));
    hipMalloc(&builder->abstract_vectors, 63 * 49 * 9 * 26 * sizeof(AbstractVector));
    hipMalloc(&builder->updates, 63 * 49 * 9 * 26 * sizeof(AbstractVector));
    hipMemset(builder->abstract_vectors, 0, 63 * 49 * 26 * 9 * sizeof(AbstractVector));
    hipHostMalloc(&builder->communication, 63 * 49 * 9 * sizeof(Vector));
    hipMalloc(&builder->opponent_ranges, 63 * 49 * 9 * sizeof(Vector));
    hipMalloc(&builder->results, 63 * 49 * 9 * sizeof(Vector));
    printf("GPU builder created\n");
    fflush(stdout);
    return builder;
}

void upload_c(Builder *builder, int index, DataType *vector) {
    memcpy(builder->communication + index, vector, 1326 * sizeof(DataType));
}

void download_c(Builder *builder, int index, DataType *vector) {
    memcpy(vector, builder->communication + index, 1326 * sizeof(DataType));
}

int build_river_cuda(long cards, DataType bet, Builder *builder) {
    hipError_t err;
    int start = builder->current_index;
    int abstract_vector_index = 0;
    int state_index = 0;
    int state_size = sizeof(State) * (28);
    State *root = (State *) malloc(state_size);

    State *device_root = builder->states + builder->current_index * 28;
    AbstractVector *abstract_vectors = builder->abstract_vectors + builder->current_index * 26;
    AbstractVector *updates = builder->updates + builder->current_index * 26;
    builder->current_index += 1;

    build_river(cards, bet, root, device_root, &state_index, abstract_vectors, updates,
                &abstract_vector_index);
    hipMemcpy(device_root, root, state_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Build error: %s\n", hipGetErrorString(err));
        fflush(stdout);
    }
//    printf("index: %d\n", start);
//    printf("vector index: %d\n", abstract_vector_index);
//    fflush(stdout);
    return start;
}
}